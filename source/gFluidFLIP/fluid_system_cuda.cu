#include "hip/hip_runtime.h"
#include "fluid_system_cuda.cuh"

#include "fluid_params.h"

#include "cutil_math.h" // cutil32.lib

__constant__ FluidParams fp;

extern "C"  __global__ void integrateParticles(float3 *pos, float3 *vel) {
  uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

  vel[i] += fp.dt * fp.gravity;
  pos[i] += vel[i] * fp.dt;
}

__global__ void handleParticleCollision(float3 *pos, float3 *vel) {
  uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

  if (pos[i].x < fp.h) {
    pos[i].x = fp.h;
    vel[i].x = 0.0f;
  } else if (pos[i].x > (fp.gridres.x - 2) * fp.h) {
    pos[i].x = (fp.gridres.x - 2) * fp.h;
    vel[i].x = 0.0f;
  }
  if (pos[i].y < fp.h) {
    pos[i].y = fp.h;
    vel[i].y = 0.0f;
  } else if (pos[i].y > (fp.gridres.y - 2) * fp.h) {
    pos[i].y = (fp.gridres.y - 2) * fp.h;
    vel[i].y = 0.0f;
  }
  if (pos[i].z < fp.h) {
    pos[i].z = fp.h;
    vel[i].z = 0.0f;
  } else if (pos[i].z > (fp.gridres.z - 2) * fp.h) {
    pos[i].z = (fp.gridres.z - 2) * fp.h;
    vel[i].z = 0.0f;
  }
}
